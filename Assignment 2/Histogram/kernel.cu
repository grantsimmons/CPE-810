#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BIN_COUNT (int) 32

template <int BINS>
__global__ void histKernel(int* a, int arr_size, int* hist, int bin_width) {

    __shared__ unsigned int hist_local[BINS];
    
    if (blockDim.x < BINS) {
        //#pragma unroll
        for (int i = 0; i < BINS; i += blockDim.x) {
            hist_local[i + threadIdx.x] = 0;
        }
    }
    else {
        if (threadIdx.x < BINS) {
            hist_local[threadIdx.x] = 0;
        }
    }
    
    __syncthreads();

    int arr_index = blockIdx.x * blockDim.x + threadIdx.x;

    if(arr_index < arr_size) {
        int bin = a[arr_index] / bin_width;
        //printf("Thread: %d, Block: %d, Dim: %d; incrementing bin %d\n", threadIdx.x, blockIdx.x, blockDim.x, bin);
        atomicAdd(&hist_local[bin], 1);
        //atomicAdd(&hist[bin], 1);
    }

    __syncthreads();

    if (blockDim.x < BINS) {
        //#pragma unroll
        for (int i = 0; i < BINS; i += blockDim.x) {
            atomicAdd(&hist[i + threadIdx.x], hist_local[i + threadIdx.x]);
        }
    }
    else {
        if (threadIdx.x < BINS) {
            atomicAdd(&hist[threadIdx.x], hist_local[threadIdx.x]);
        }
    }
}


float calculateHist(int* a, int arr_size, int* hist, int bin_width, int block_width) {
    int* dev_a;
    int* dev_hist;

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkCudaErrors(hipSetDevice(0));

    // Allocate GPU buffers for three vectors (two input, one output)    .
    checkCudaErrors(hipMalloc((void**)&dev_a, arr_size * sizeof(int)));

    checkCudaErrors(hipMalloc((void**)&dev_hist, BIN_COUNT * sizeof(int)));

    // Copy input vectors from host memory to GPU buffers.
    checkCudaErrors(hipMemcpy(dev_a, a, arr_size * sizeof(int), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(dev_hist, hist, BIN_COUNT * sizeof(int), hipMemcpyHostToDevice));

    dim3 threads(block_width, 1, 1);

    dim3 grid( ((arr_size - 1) / threads.x) + 1, 1, 1 );
    //dim3 grid(1, 1, 1);

    float time = 0;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
    
    histKernel<BIN_COUNT> << <grid, threads >> > (dev_a, arr_size, dev_hist, bin_width);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));
    printf("Time to generate: %3.5f ms \n", time);

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(hist, dev_hist, BIN_COUNT * sizeof(int), hipMemcpyDeviceToHost));

    return time;
}

int main(int argc, char** argv)
{
    printf("[Histogram computation Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char**)argv, "help") ||
        checkCmdLineFlag(argc, (const char**)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -bin_count=BinCount -vec_dim=VecDim ()\n");

        exit(EXIT_SUCCESS);
    }

    long int vec_dim_t = 100000000;
    int block_width = 256;

    if (checkCmdLineFlag(argc, (const char**)argv, "vec_dim"))
        vec_dim_t = getCmdLineArgumentInt(argc, (const char**)argv, "vec_dim");
    if (checkCmdLineFlag(argc, (const char**)argv, "block_width"))
        block_width = getCmdLineArgumentInt(argc, (const char**)argv, "block_width");
    //if (checkCmdLineFlag(argc, (const char**)argv, "vec_dim"))
    //    bin_count_t = getCmdLineArgumentInt(argc, (const char**)argv, "vec_dim");

    const long int vec_dim = vec_dim_t;

    srand(time(NULL));

    int* a = new int[vec_dim];

    const int max_value = RAND_MAX;
    //const int max_value = 2;

    const int bin_width = (int)((max_value - 1) / BIN_COUNT) + 1;
    printf("Bin width: %d\n", bin_width);

    printf("Array values: ");
    for (long int i = 0; i < vec_dim; i++) {
        int uval = (int)rand();
        //int uval = 20;
        int val = uval % max_value;
        a[i] = val;
        //printf("%d, ", val);
    }

    printf("\n");

    int* hist = new int[BIN_COUNT];

    for (int i = 0; i < BIN_COUNT; i++) {
        hist[i] = 0;
    }

    float time = calculateHist(a, vec_dim, hist, bin_width, block_width);

    printf("Array Size: %d, Bin Count: %d\n", vec_dim, BIN_COUNT);
    for (int i = 0; i < BIN_COUNT; i++) {
        printf("%d, ", hist[i]);
    }
    printf("\n");

    int block_count = (((vec_dim - 1) / block_width) + 1);
    int total_global_atomics = BIN_COUNT * block_count;
    int total_shared_atomics = vec_dim;

    printf("Global Memory Reads: %d\n", vec_dim);
    printf(" (Including atomic operations): %d\n", vec_dim + total_global_atomics);
    printf(" (Per Element): %f\n", (float)(vec_dim + total_global_atomics) / vec_dim);
    printf("Shared Memory Writes: %d\n", vec_dim);
    printf(" (Per Block): %.1f\n", (float) (vec_dim / block_width));
    printf("Global Memory Writes: %d\n", total_global_atomics);
    printf(" (Per Element): %f\n", (float)(total_global_atomics) / vec_dim);
    printf("GFLOPS: %f\n", (float) (vec_dim / (time / 1000)) / 1000000000);

    checkCudaErrors(hipDeviceReset());

    return 0;
}
